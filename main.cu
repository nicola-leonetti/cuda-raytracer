#include <time.h>

#include "camera.h"
#include "color.h"
#include "common.h"
#include "material.h"
#include "sphere.h"

#define NUMBER_OF_SPHERES 489

__host__ inline void print_device_info(int id) {
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    
    fprintf(stderr, "\n-----------------Device properties-----------------\n");
    fprintf(stderr, "GPU name: %s \n", properties.name);
    fprintf(
        stderr, 
        "Compute capability: %d.%d\n", 
        properties.major,
        properties.minor
    );
    fprintf(
        stderr, 
        "Max number of threads per block: %d \n", 
        properties.maxThreadsPerBlock
    );
    fprintf(
        stderr, 
        "Max size of a block of threads: (%d, %d, %d) \n", 
        properties.maxThreadsDim[0],
        properties.maxThreadsDim[1],
        properties.maxThreadsDim[2]
    );
    fprintf(
        stderr, 
        "Max size of grid of blocks: (%d, %d, %d) \n",
        properties.maxGridSize[0],
        properties.maxGridSize[1],
        properties.maxGridSize[2]
    );
    fprintf(stderr, "---------------------------------------------------\n\n");
}

__host__ double cpuSecond() {
    struct timespec ts;
    timespec_get(&ts, TIME_UTC);
    return ((double) ts.tv_sec + (double) ts.tv_nsec * 1.e-9);
}

// TODO Vedere se riesco a fare in modo che inizializzi tutto su device
// magari parallelizzando
__host__ void init_world(t_sphere world[]) {
    fprintf(stderr, "Initializing spheres...");

    // Ground sphere (Lambertian material)
    world[0] = sphere_new(
        point3_new(0, -1000, 0), 1000, new_lambertian(COLOR_GRAY));

    world[1] = sphere_new(point3_new(0, 1, 0), 1.0, new_dielectric(1.5));
    world[2] = \
        sphere_new(point3_new(-4, 1, 0), 1.0, new_lambertian(COLOR_BLUE));
    world[3] = \
        sphere_new(point3_new(4, 1, 0), 1.0, new_metal(COLOR_GREEN, 0.0));

    // Create a grid of random spheres
    int index = 4;
    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            // Randomize material choice
            my_decimal choose_mat = h_random_my_decimal();
            t_point3 center = point3_new(a + 0.9 * h_random_my_decimal(), 0.2, b + 0.9 * h_random_my_decimal());
            t_material sphere_material;

            if (choose_mat < 0.8) {
                // Lambertian (diffuse)
                sphere_material = new_lambertian(
                    color_new(h_random_my_decimal()*h_random_my_decimal(), 
                              h_random_my_decimal()*h_random_my_decimal(), 
                              h_random_my_decimal()*h_random_my_decimal()
                            ));
                world[index++] = sphere_new(center, 0.2, sphere_material);
            } 
            else if (choose_mat < 0.95) {
                // Metal
                t_color color = color_new(h_random_my_decimal_in(0.5, 1), h_random_my_decimal_in(0.5, 1), h_random_my_decimal_in(0.5, 1));
                sphere_material = new_metal(color, h_random_my_decimal_in(0, 0.5));
                world[index++] = sphere_new(center, 0.2, sphere_material);
            } 
            else {
                // Dielectric (glass)
                world[index++] = sphere_new(center, 0.2, new_dielectric(1.5));
            }
            
        }
    }

    fprintf(stderr, "\r                            \r");
    fprintf(stderr, "Spheres initilized\n");
}

__host__ void write_PPM_img_to_stdout(unsigned char *img, int width, int height) {
    // PPM header
    printf("P3\n%d %d\n255\n", width, height);
    for (int pixel = 0; pixel < width*height*3; pixel+=3) {
        printf("%d %d %d\n", img[pixel], img[pixel+1], img[pixel+2]);
    }
}

__host__ int main() {

    print_device_info(0);

    srand((unsigned int) RNG_SEED);

    // Initialize spheres on host
    int world_size = NUMBER_OF_SPHERES*sizeof(t_sphere);
    t_sphere *h_world = (t_sphere*) malloc(world_size); 
    init_world(h_world);

    // Copy spheres host -> device
    t_sphere *d_world;
    CHECK(hipMalloc((void**)&d_world, world_size));
    CHECK(hipMemcpy(d_world, h_world, world_size, hipMemcpyHostToDevice));

    // TODO Vedere se possibile creare oggetto direttamente su device per
    // ottimizzare
    // Create camera object and move it on device
    t_camera cam = camera_new(ASPECT_RATIO, VIEWPORT_WIDTH, 
                              VERTICAL_FOV_DEGREES, (t_point3) LOOK_FROM, 
                              (t_point3) LOOK_AT, DEFOCUS_ANGLE, 
                              FOCUS_DISTANCE);
    t_camera *h_cam = &cam;
    t_camera *d_cam;
    CHECK(hipMalloc((void**)&d_cam, sizeof(cam)));
    CHECK(hipMemcpy(d_cam, h_cam, sizeof(cam), hipMemcpyHostToDevice));

    // Allocate on device one RNG state for each pixel
    int number_of_pixels = cam.image_width*cam.image_height;
    hiprandState *d_random_states;
    CHECK(hipMalloc(
        (void**) &d_random_states, 
        number_of_pixels*sizeof(hiprandState)
    ));

    // Allocate space for the image on host and device
    long img_size = cam.image_height*cam.image_width*sizeof(unsigned char)*3;
    unsigned char *h_result_img = (unsigned char*) malloc(img_size);
    unsigned char *d_result_img;
    CHECK(hipMalloc((void**)&d_result_img, img_size));
    CHECK(hipMemcpy(d_result_img, h_result_img, img_size, hipMemcpyHostToDevice));

    fprintf(
        stderr,
        "Launching render kernel with 2D grid shape (%u, %u)\n", 
        (cam.image_width + block.x - 1) / block.x, 
        (cam.image_height + block.y - 1) / block.y
    );
        
    double start = cpuSecond();

    dim3 grid(
        (cam.image_width + block.x - 1) / block.x, 
        (cam.image_height + block.y - 1) / block.y
    );
    camera_render<<<grid, block>>>(
        d_cam, 
        d_world, 
        NUMBER_OF_SPHERES, 
        d_result_img, 
        d_random_states
    );
    hipDeviceSynchronize();

    double end = cpuSecond();
    fprintf(stderr, "Computation time: %.6fs\n", end - start);

    // Copy the result image device -> host -> .ppm file 
    fprintf(
        stderr, 
        "Image size: %dx%d, %d channels, %ld bytes\n", 
        cam.image_height, 
        cam.image_width,
        3,
        img_size
    );
    CHECK(hipMemcpy(h_result_img, d_result_img, img_size, hipMemcpyDeviceToHost));
    write_PPM_img_to_stdout(h_result_img, cam.image_width, cam.image_height);

    CHECK(hipFree(d_world));
    CHECK(hipFree(d_cam));
    CHECK(hipFree(d_result_img));
    free(h_world);
    free(h_result_img);
}
